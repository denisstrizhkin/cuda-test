#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <vector>

#include "cpu.h"
#include "mat.cuh"

template <typename T> void test_attention_cpu() {
  const size_t N = 3;
  const size_t D = 2;
  const std::vector<T> q = {
      1.0, 0.0, 0.0, 1.0, 1.0, 1.0,
  };
  const std::vector<T> k = {
      1.0, 0.0, 0.0, 1.0, 0.0, 0.0,
  };
  const std::vector<T> v = {
      1.0, 2.0, 3.0, 4.0, 5.0, 6.0,
  };
  const auto o = cpu::attention<T, N, D>(q, k, v);
  const std::vector<T> o_expected = {
      2.27158774, 3.27159260, 2.99901458, 4.00001944, 2.46607088, 3.46607088,
  };
  assert(o.size() == N * D);
  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < D; j++) {
      const auto k = i * D + j;
      const auto got = o[k];
      const auto expected = o_expected[k];
      const auto diff = std::abs(expected - got);
      if (diff > 1e-3) {
        std::cout << "at (" << i << "," << j << ") expected: " << expected
                  << ", got: " << got << "\n";
        assert(false);
      }
    }
  }
}

void test_gpu() {
  const size_t N = 4;
  const size_t D = 8;
  const auto mQ = cpu::random<float, N, D>();
  const auto mK = cpu::random<float, N, D>();
  const auto mV = cpu::random<float, N, D>();
  const auto result = cpu::attention<float, N, D>(mQ, mK, mV);
  assert(result.size() == N * D);
  const auto gMQ = Mat::Mat<float, N, D>(mQ);
  const auto gMK = Mat::Mat<float, N, D>(mK);
  const auto gMV = Mat::Mat<float, N, D>(mV);
  const auto gResult = Mat::Mat<float, N, D>::attention(gMQ, gMK, gMV);
  assert(gResult.size() == N * D);
  for (auto i = 0; i < N; i++) {
    for (auto j = 0; j < D; j++) {
      const auto got = gResult.at(i ,j);
      const auto expected = result[i * D + j];
      const auto diff = std::abs(expected - got);
      if (diff > 1e-3) {
        std::cout << "at (" << i << "," << j << ") expected: " << expected
                  << ", got: " << got << "\n";
        assert(false);
      }
    }
  }
}

int main(void) {
  test_attention_cpu<float>();
  test_gpu();
  return EXIT_SUCCESS;
}
