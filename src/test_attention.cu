#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <vector>

#include "cpu.h"
#include "mat.cuh"

template <typename T, size_t N, size_t M>
void compare(const Mat::Mat<T, N, M> &m, const std::vector<T> &v) {
  const size_t size = N * M;
  const size_t v_size = v.size();
  if (v_size != size) {
    std::cout << "vector size: expected: " << size << ", got: " << v_size
              << "\n";
    assert(false);
  }
  const size_t m_size = m.size();
  if (m_size != size) {
    std::cout << "mat size: expected: " << size << ", got: " << m_size << "\n";
    assert(false);
  }
  const size_t m_rows = m.rows();
  if (m_rows != N) {
    std::cout << "mat rows: expected: " << N << ", got: " << m_rows << "\n";
    assert(false);
  }
  const size_t m_cols = m.cols();
  if (m_cols != M) {
    std::cout << "mat cols: expected: " << M << ", got: " << m_cols << "\n";
    assert(false);
  }
  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < M; j++) {
      const T got = m.at(i, j);
      const T expected = v[i * M + j];
      const T diff = std::abs(expected - got);
      if (diff > 1e-6) {
        std::cout << "at (" << i << "," << j << ") expected: " << expected
                  << ", got: " << got << "\n";
        assert(false);
      }
    }
  }
}

template <typename T, size_t N, size_t M> void test_from_vector() {
  const auto v = cpu::random<T, N, M>();
  const auto m = Mat::Mat<T, N, M>(v);
  compare(m, v);
}

template <typename T, size_t N, size_t M> void test_transpose() {
  const auto a = cpu::random<T, N, M>();
  const auto aT = cpu::transpose<T, N, M>(a);
  const auto mA = Mat::Mat<T, N, M>(a);
  const auto mAT = mA.transpose();
  compare(mAT, aT);
}

template <typename T, size_t N, size_t M> void test_softmax() {
  const auto a = cpu::random<T, N, M>();
  const auto smaxA = cpu::softmax<T, N, M>(a);
  const auto mA = Mat::Mat<T, N, M>(a);
  const auto smaxMA = mA.softmax();
  compare(smaxMA, smaxA);
}

template <typename T, size_t N, size_t M> void test_attention() {
  const auto q = cpu::random<T, N, M>();
  const auto k = cpu::random<T, N, M>();
  const auto v = cpu::random<T, N, M>();
  const auto o = cpu::attention<T, N, M>(q, k, v);
  const auto mQ = Mat::Mat<T, N, M>(q);
  const auto mK = Mat::Mat<T, N, M>(k);
  const auto mV = Mat::Mat<T, N, M>(v);
  const auto mO = Mat::Mat<T, N, M>::attention(mQ, mK, mV);
  compare(mO, o);
}

template <typename T> void test_attention_cpu() {
  const size_t N = 3;
  const size_t D = 2;
  const std::vector<T> q = {
      1.0, 0.0, 0.0, 1.0, 1.0, 1.0,
  };
  const std::vector<T> k = {
      1.0, 0.0, 0.0, 1.0, 0.0, 0.0,
  };
  const std::vector<T> v = {
      1.0, 2.0, 3.0, 4.0, 5.0, 6.0,
  };
  const auto o = cpu::attention<T, N, D>(q, k, v);
  const std::vector<T> o_expected = {
      2.27158774, 3.27159260, 2.99901458, 4.00001944, 2.46607088, 3.46607088,
  };
  assert(o.size() == N * D);
  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < D; j++) {
      const auto k = i * D + j;
      const auto got = o[k];
      const auto expected = o_expected[k];
      const auto diff = std::abs(expected - got);
      if (diff > 1e-3) {
        std::cout << "at (" << i << "," << j << ") expected: " << expected
                  << ", got: " << got << "\n";
        assert(false);
      }
    }
  }
}

int main(void) {
  test_attention_cpu<float>();
  const size_t N = 8;
  const size_t D = 4;
  test_from_vector<float, N, D>();
  test_transpose<float, N, D>();
  test_softmax<float, N, D>();
  test_attention<float, N, N>();
  test_from_vector<float, D, N>();
  test_transpose<float, D, N>();
  test_softmax<float, D, N>();
  test_attention<float, D, N>();
  return EXIT_SUCCESS;
}
