#include "mat.cuh"
#include "rand.h"
#include "cpu.h"

#include <cassert>
#include <chrono>
#include <vector>

void print_bench_header(const std::string &name, const size_t n, const size_t m,
                        const size_t k) {
  std::cout << "\n=== bench " << name << " (" << n << "x" << m << ") by (" << m
            << "x" << k << ") mul ===\n";
}

void print_bench_footer(const float millis) {
  std::cout << "=== finished in " << millis << "ms ===\n";
}

template <typename T, size_t N, size_t M, size_t K, typename DotProductFunc>
void bench_mul(const std::string &test_name, DotProductFunc dot_func) {
  print_bench_header(test_name, N, M, K);
  const auto a = Mat::Mat<T, N, M>::random();
  const auto b = Mat::Mat<T, M, K>::random();
  hipEvent_t start_event, stop_event;
  CUDA_CHECK(hipEventCreate(&start_event));
  CUDA_CHECK(hipEventCreate(&stop_event));
  CUDA_CHECK(hipEventRecord(start_event));
  const auto c = (a.*dot_func)(b);
  CUDA_CHECK(hipEventRecord(stop_event));
  CUDA_CHECK(hipEventSynchronize(stop_event));
  float millis = 0;
  CUDA_CHECK(hipEventElapsedTime(&millis, start_event, stop_event));
  print_bench_footer(millis);
}

template <typename T, size_t N, size_t M, size_t K> void bench_mul_naive() {
  bench_mul<T, N, M, K>("naive", &Mat::Mat<T, N, M>::template dot_naive<K>);
}

template <typename T, size_t N, size_t M, size_t K> void bench_mul_shared() {
  bench_mul<T, N, M, K>("shared", &Mat::Mat<T, N, M>::template dot_shared<K>);
}

template <typename T, size_t N, size_t M, size_t K>
void bench_mul_shared_with_warp_intrinsics() {
  bench_mul<T, N, M, K>(
      "shared with warp intrinsics",
      &Mat::Mat<T, N, M>::template dot_shared_with_warp_intrinsics<K>);
}

template <typename T, size_t N, size_t M, size_t K> void bench_mul_cpu() {
  print_bench_header("cpu", N, M, K);
  const auto a = cpu::random<T, N, M>();
  const auto b = cpu::random<T, M, K>();
  const auto start_time = std::chrono::high_resolution_clock::now();
  const auto c = cpu::dot<T, N, M, K>(a, b);
  const auto end_time = std::chrono::high_resolution_clock::now();
  const std::chrono::duration<float, std::milli> duration =
      end_time - start_time;
  const auto millis = duration.count();
  print_bench_footer(millis);
}

int main(void) {
  hipDeviceProp_t deviceProp;
  CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
  std::cout << "=== Device Info: " << deviceProp.name << " ===\n"
            << "  Compute Capability: " << deviceProp.major << "."
            << deviceProp.minor << "\n"
            << "  Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024)
            << "MB\n"
            << "  Memory Clock Rate: " << deviceProp.memoryClockRate / 1000.0
            << "MHz (Effective)\n"
            << "  GPU Clock Rate (Max): " << deviceProp.clockRate / 1000.0
            << "MHz\n"
            << "  Shared Memory per Block: "
            << deviceProp.sharedMemPerBlock / 1024.0 << "KB\n"
            << "  Shared Memory per Multiprocessor: "
            << deviceProp.sharedMemPerMultiprocessor / 1024.0 << "\n"
            << "  Registers per Block: " << deviceProp.regsPerBlock << "\n"
            << "  Max Threads per Block: " << deviceProp.maxThreadsPerBlock
            << "\n"
            << "  Max Threads per Multiprocessor: "
            << deviceProp.maxThreadsPerMultiProcessor << "\n"
            << "  Multiprocessor Count: " << deviceProp.multiProcessorCount
            << "\n"
            << "  L2 Cache Size: " << deviceProp.l2CacheSize / 1024.0 << "KB"
            << "\n"
            << "===\n";
  std::cout << "\n--- Benchmarking CPU Multiplication ---\n";
  bench_mul_cpu<float, 8, 16, 32>();
  bench_mul_cpu<float, 64, 128, 256>();
  // bench_mul_cpu<float, 1024, 2048, 4096>(); // too slow anyway
  std::cout << "\n--- Benchmarking Naive Multiplication ---\n";
  bench_mul_naive<float, 8, 16, 32>();
  bench_mul_naive<float, 64, 128, 256>();
  bench_mul_naive<float, 1024, 2048, 4096>();
  std::cout << "\n--- Benchmarking Shared Multiplication ---\n";
  bench_mul_shared<float, 8, 16, 32>();
  bench_mul_shared<float, 64, 128, 256>();
  bench_mul_shared<float, 1024, 2048, 4096>();
  std::cout
      << "\n--- Benchmarking Shared Multiplication With Warp Intrinsics ---\n";
  bench_mul_shared_with_warp_intrinsics<float, 8, 16, 32>();
  bench_mul_shared_with_warp_intrinsics<float, 64, 128, 256>();
  bench_mul_shared_with_warp_intrinsics<float, 1024, 2048, 4096>();
  return EXIT_SUCCESS;
}
